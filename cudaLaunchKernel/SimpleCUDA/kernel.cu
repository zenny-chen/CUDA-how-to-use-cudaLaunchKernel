﻿
#include "hip/hip_runtime.h"


#include <cstdio>

static constexpr auto arraySize = 1152U;

static __global__ void addKernel(int c[], const int a[], const int b[], int constValue)
{
    auto const gtid = threadIdx.x + blockDim.x * blockIdx.x;
    if (gtid >= arraySize) {
        return;
    }

    c[gtid] = a[gtid] * a[gtid] + (b[gtid] - constValue);
}

static void AddWithCUDATest(void)
{
    puts("======== The following is Add-With-CUDA Test ========");

    int a[arraySize];
    int b[arraySize];
    int c[arraySize] = {  };

    hipFuncAttributes attrs{ };
    auto cudaStatus = hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(addKernel));
    if (cudaStatus != hipSuccess)
    {
        printf("hipFuncGetAttributes call failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    auto const maxThreadCount = attrs.maxThreadsPerBlock;

    for (unsigned i = 0U; i < arraySize; ++i)
    {
        a[i] = i + 1;
        b[i] = a[i] * 10;
    }

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    do
    {
        cudaStatus = hipMalloc(&dev_c, sizeof(c));
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed for dev_c: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMalloc(&dev_a, sizeof(a));
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed for dev_a: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMalloc(&dev_b, sizeof(b));
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed for dev_b: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(dev_a, a, sizeof(a), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed for dev_a: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(dev_b, b, sizeof(b), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed for dev_a: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        auto const blockSize = (arraySize + maxThreadCount - 1) / maxThreadCount;

        // Launch a kernel on the GPU with one thread for each element.
        int constValue = 100;
        
        //addKernel <<< blockSize, maxThreadCount >>> (dev_c, dev_a, dev_b, constValue);
        void* args[]{ &dev_c, &dev_a, &dev_b, &constValue };
        cudaStatus = hipLaunchKernel(reinterpret_cast<const void*>(addKernel), dim3(blockSize, 1U, 1U), dim3(maxThreadCount, 1U, 1U), args, 0U, nullptr);

        if (cudaStatus != hipSuccess)
        {
            printf("hipLaunchKernel failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed for dev_c: %s\n", hipGetErrorString(cudaStatus));
            break;
        }
        // Verify the result
        bool success = true;
        for (unsigned i = 0; i < arraySize; ++i)
        {
            const int correctValue = a[i] * a[i] + (b[i] - constValue);
            if (c[i] != correctValue)
            {
                printf("Result error @%u, destination is: %d, correct value is: %d\n", i, c[i], correctValue);
                success = false;
                break;
            }
        }
        if (success) {
            puts("Result is correct!");
        }
    }
    while (false);

    if (dev_a != nullptr) {
        hipFree(dev_a);
    }
    if (dev_b != nullptr) {
        hipFree(dev_b);
    }
    if (dev_c != nullptr) {
        hipFree(dev_c);
    }
}

int main(int argc, const char* argv[])
{
    auto cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        printf("hipSetDevice call failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    AddWithCUDATest();

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        printf("hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    return 0;
}
